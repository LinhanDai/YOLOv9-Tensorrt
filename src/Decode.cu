/**
* Website: https://github.com/LinhanDai
* @author dailinhan
* @date 24-02-23 10:24
                   _ooOoo_
                  o8888888o
                  88" . "88
                  (| -_- |)
                  O\  =  /O
               ____/`---'\____
             .'  \\|     |//  `.
            /  \\|||  :  |||//  \
           /  _||||| -:- |||||-  \
           |   | \\\  -  /// |   |
           | \_|  ''\---/''  |   |
           \  .-\__  `-`  ___/-. /
         ___`. .'  /--.--\  `. . __
      ."" '<  `.___\_<|>_/___.'  >'"".
     | | :  `- \`.;`\ _ /`;.`/ - ` : | |
     \  \ `-.   \_ __\ /__ _/   .-` /  /
======`-.____`-.___\_____/___.-`____.-'======
                   `=---='
^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^
            no error       no bug
*/

#include <hip/hip_runtime.h>
#include <cstdio>


static __global__ void transpose_kernel(float *src, int num_bboxes, int num_elements,float *dst, int edge)
{
    int position = blockDim.x * blockIdx.x + threadIdx.x;
    if (position>=edge)
        return;
    dst[position]=src[position / num_elements + (position % num_elements) * num_bboxes];
}

extern "C" void transpose_kernel_invoker(float *src, int num_bboxes, int num_elements,float *dst,hipStream_t stream)
{
    int edge = num_bboxes * num_elements;
    int block = 256;
    int gird = ceil(edge / (float)block);
    transpose_kernel<<<gird,block,0,stream>>>(src,num_bboxes,num_elements, dst, edge);
}

static __device__ void affine_project(float* matrix, float x, float y, float* ox, float* oy)
{
    *ox = matrix[0] * x + matrix[1] * y + matrix[2];
    *oy = matrix[3] * x + matrix[4] * y + matrix[5];
}

static __global__ void decode_kernel(
        float* predict, int num_bboxes, int num_classes,
        float confidence_threshold, float* invert_affine_matrix,
        float* parray, int max_objects, int NUM_BOX_ELEMENT)
{

    int position = blockDim.x * blockIdx.x + threadIdx.x;
    if (position >= num_bboxes) return;

    float* pitem     = predict + (4 + num_classes) * position;

    float* class_confidence = pitem + 4;
    float confidence        = *class_confidence++;
    int label               = 0;
    for(int i = 1; i < num_classes; ++i, ++class_confidence)
    {
        if(*class_confidence > confidence)
        {
            confidence = *class_confidence;
            label      = i;
        }
    }

    // confidence *= objectness;
    if(confidence < confidence_threshold)
        return;

    int index = atomicAdd(parray, 1);
    if(index >= max_objects)
        return;
    // printf("index %d max_objects %d\n", index,max_objects);
    float cx         = pitem[0];
    float cy         = pitem[1];
    float width      = pitem[2];
    float height     = pitem[3];

    float left   = cx - width * 0.5f;
    float top    = cy - height * 0.5f;
    float right  = cx + width * 0.5f;
    float bottom = cy + height * 0.5f;

    affine_project(invert_affine_matrix, left,  top,    &left,  &top);
    affine_project(invert_affine_matrix, right, bottom, &right, &bottom);


    float* pout_item = parray + 1 + index * NUM_BOX_ELEMENT;
    *pout_item++ = left;
    *pout_item++ = top;
    *pout_item++ = right;
    *pout_item++ = bottom;
    *pout_item++ = confidence;
    *pout_item++ = label;
    *pout_item++ = 1; // 1 = keep, 0 = ignore
}

static __device__ float box_iou(
        float aleft, float atop, float aright, float abottom,
        float bleft, float btop, float bright, float bbottom)
{
    float cleft 	= max(aleft, bleft);
    float ctop 		= max(atop, btop);
    float cright 	= min(aright, bright);
    float cbottom 	= min(abottom, bbottom);

    float c_area = max(cright - cleft, 0.0f) * max(cbottom - ctop, 0.0f);
    if(c_area == 0.0f)
        return 0.0f;

    float a_area = max(0.0f, aright - aleft) * max(0.0f, abottom - atop);
    float b_area = max(0.0f, bright - bleft) * max(0.0f, bbottom - btop);
    return c_area / (a_area + b_area - c_area);
}

static __global__ void fast_nms_kernel(float* bboxes,
                                       int max_objects,
                                       float threshold,
                                       int NUM_BOX_ELEMENT)
{
    int position = (blockDim.x * blockIdx.x + threadIdx.x);
    int count = min((int)*bboxes, max_objects);
    if (position >= count)
        return;

    // left, top, right, bottom, confidence, class, keepflag
    float* pcurrent = bboxes + 1 + position * NUM_BOX_ELEMENT;
    for(int i = 0; i < count; ++i)
    {
        float* pitem = bboxes + 1 + i * NUM_BOX_ELEMENT;
        if(i == position || pcurrent[5] != pitem[5]) continue;

        if(pitem[4] >= pcurrent[4])
        {
            if(pitem[4] == pcurrent[4] && i < position)
                continue;

            float iou = box_iou(
                    pcurrent[0], pcurrent[1], pcurrent[2], pcurrent[3],
                    pitem[0],    pitem[1],    pitem[2],    pitem[3]
            );

            if(iou > threshold)
            {
                pcurrent[6] = 0;  // 1=keep, 0=ignore
                return;
            }
        }
    }
}

extern "C" void decode_kernel_invoker(
        float* predict, int num_bboxes, int num_classes, float confidence_threshold,
        float nms_threshold, float* invert_affine_matrix, float* parray, int max_objects,
        int num_box_element, hipStream_t stream)
{
    auto block = num_bboxes > 512 ? 512 : num_bboxes;
    auto grid = (num_bboxes + block - 1) / block;
    decode_kernel<<<grid, block, 0, stream>>>(
            predict, num_bboxes, num_classes,
            confidence_threshold, invert_affine_matrix,
            parray, max_objects, num_box_element);

    block = max_objects > 512 ? 512 : max_objects;
    grid = (max_objects + block - 1) / block;
    fast_nms_kernel<<<grid, block, 0, stream>>>(parray, max_objects, nms_threshold, num_box_element);
}